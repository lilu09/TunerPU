
#include <iostream>
#include <assert.h>
#include <lib/tuneit.h>
#include <lib/vector_scale_tunable.h>

using namespace std;

int main()
{

	assert( hipDeviceReset() == hipSuccess ) ;



	size_t const depth=2;
	vector<bool> const mask(2,true);

	tuneit::tuneit_settings<VECTOR_SCALE_NUM_DIM> st{depth, mask, true, false, true, 40, { {1,200} } };

	constexpr size_t num_vertices=2;

	tuneit::tuneit< VECTOR_SCALE_NUM_VARIANTS, num_vertices, vector_scale_tunable<float, size_t>,
			float, size_t> mytuner(st);

	mytuner.train();

	cout<<"prediction is: "<<mytuner.predict(20)<<endl;;



	return EXIT_SUCCESS;
}
