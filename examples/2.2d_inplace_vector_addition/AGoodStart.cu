
#include <iostream>
#include <assert.h>
#include <lib/tuneit.h>
#include <lib/matrix_vector_mul_tunable.h>

using namespace std;

int main()
{

	assert( hipDeviceReset() == hipSuccess ) ;



	size_t const depth=2;
	vector<bool> const mask(2,true);

	tuneit::tuneit_settings<MATRIX_VECTOR_MUL_NUM_DIM> st{depth, mask, true, false, true, 40, { {1,200}, {1,200} } };

	constexpr size_t num_vertices=4;

	tuneit::tuneit< MATRIX_VECTOR_MUL_NUM_VARIANTS, num_vertices, matrix_vector_mul_tunable<float, size_t, size_t>,
			float, size_t, size_t> mytuner(st);

	mytuner.train();

	cout<<"prediction is: "<<mytuner.predict(20,20)<<endl;;



	return EXIT_SUCCESS;
}
