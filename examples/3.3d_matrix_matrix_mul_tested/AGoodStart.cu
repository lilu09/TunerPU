
#include <iostream>
#include <assert.h>

#include <bits/stdc++.h>

/*using namespace std;*/

/*#include <lib/vectorpu.h>*/
#include <lib/matrix_mul_tunable.h>
#include <tuneit.h>
#include <meterpu.h>

#define OPT
#define BASE


//#include <hip/hip_runtime.h>
//#include <hipblas.h>

/*void matrix_mul_cublas(float const * const a,float const * const b,float * const c,size_t const ha,size_t const wa,size_t const wb)*/
/*{*/

/*const float alf = 1.0f;*/
/*const float bet = 0.0f;*/
/*const float *alpha = &alf;*/
/*const float *beta = &bet;*/

/*hipblasStatus_t stat;*/
/*hipblasHandle_t handle;*/

/*stat = hipblasCreate(&handle);*/
/*assert(stat == HIPBLAS_STATUS_SUCCESS);*/

/*hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, static_cast<int>(wb), static_cast<int>(ha), static_cast<int>(wa), alpha, b, static_cast<int>(wb), a, static_cast<int>(wa), beta, c, static_cast<int>(wb));*/

/*hipblasDestroy(handle);*/
/*}*/

//8
int main(int argc, char* argv[])
{

	assert( hipDeviceReset() == hipSuccess );

	assert( argc == 2 );

	unsigned int const problem_size=std::atoi(argv[1]);


	/*const size_t ha=200, wa=200, wb=200;*/
	const size_t ha=problem_size, wa=problem_size, wb=problem_size;

/*#ifdef BASE*/

/*#endif*/

/*#ifdef OPT*/
	{



		tuneit::tuneit_settings<MATRIX_MUL_NUM_DIM, MATRIX_MUL_NUM_VARIANTS> st{2, std::vector<bool>(4,true), true, false, true, 5, {{1,10}, {1,10}, {1,10}} };

		tuneit::tuneit< MATRIX_MUL_NUM_VARIANTS, 8, matrix_mul_tunable<float, size_t, size_t, size_t>, float, size_t, size_t, size_t> mytuner(st);

		std::cout<<"before train"<<std::endl;

		mytuner.train();

		std::cout<<"after train"<<std::endl;

		vectorpu::vector<float> a(wa*ha,1), b(wa*wb,1), c(ha*wb,0);

		std::cout<<"before run"<<std::endl;

		mytuner.run(mytuner.predict(ha,wa,wb), a, b, c, ha, wa, wb);

		std::cout<<"after run"<<std::endl;

		using namespace meterpu;
		meterpu::meter<meterpu::CPU_Time> my_meter;

		for(size_t i=0;i<3;++i){
			std::cerr<<"Testing opt, problem size: "<<problem_size<<", No."<<i<<std::endl;
			my_meter.start();

			mytuner.run(mytuner.predict(ha,wa,wb), a, b, c, ha, wa, wb);

			my_meter.stop();
			my_meter.calc();
			std::cout<<my_meter.get_value()<<std::endl;
		}

		/*std::cout<<mytuner.predict(ha,wa,wb)<<std::endl;*/


		/*std::for_each(RI(c), REI(c), [](float const i){assert(i==200.0f);}) ;*/
		/*std::for_each(RI(c), REI(c), [](float const i){assert(i==10.0f);}) ;*/
		/*std::for_each(RI(c), REI(c), [problem_size](float const i){assert(i==float(problem_size));}) ;*/

	}
/*#endif*/




	return EXIT_SUCCESS;
}
