
#include <iostream>
#include <assert.h>

#include <bits/stdc++.h>

/*using namespace std;*/

/*#include <lib/vectorpu.h>*/
#include <lib/matrix_mul_tunable.h>
#include <tuneit.h>
#include <meterpu.h>

#define OPT
#define BASE


int main(int argc, char* argv[])
{

	assert( hipDeviceReset() == hipSuccess );

	assert( argc == 2 );

	unsigned int const problem_size=std::atoi(argv[1]);


	const size_t ha=problem_size, wa=problem_size, wb=problem_size;

	{



		tuneit::tuneit_settings<MATRIX_MUL_NUM_DIM, MATRIX_MUL_NUM_VARIANTS> st{2, std::vector<bool>(4,true), true, false, true, 5, {{1,10}, {1,10}, {1,10}} };

		tuneit::tuneit< MATRIX_MUL_NUM_VARIANTS, 8, matrix_mul_tunable<float, size_t, size_t, size_t>, float, size_t, size_t, size_t> mytuner(st);

		std::cout<<"before train"<<std::endl;

		mytuner.train();

		std::cout<<"after train"<<std::endl;

		vectorpu::vector<float> a(wa*ha,1), b(wa*wb,1), c(ha*wb,0);

		std::cout<<"before run"<<std::endl;

		mytuner.run(mytuner.predict(ha,wa,wb), a, b, c, ha, wa, wb);

		std::cout<<"after run"<<std::endl;

		using namespace meterpu;
		meterpu::meter<meterpu::CPU_Time> my_meter;

		for(size_t i=0;i<3;++i){
			std::cerr<<"Testing opt, problem size: "<<problem_size<<", No."<<i<<std::endl;
			my_meter.start();

			mytuner.run(mytuner.predict(ha,wa,wb), a, b, c, ha, wa, wb);

			my_meter.stop();
			my_meter.calc();
			std::cout<<my_meter.get_value()<<std::endl;
		}


	}




	return EXIT_SUCCESS;
}
